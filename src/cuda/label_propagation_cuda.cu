#include "hip/hip_runtime.h"
#include "label_propagation_cuda.hpp"
#include <algorithm>
#include <iostream>
#include <stdexcept>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <type_traits>
#include <vector>

namespace {

constexpr int MAX_LABELS = 10; // must cover possible label values
constexpr int MAX_CUDA_BLOCK_SIZE = 1024;

__global__ void
update_edge_labels_kernel(const Hypergraph::VertexId* edge_vertices, const std::size_t* edge_offsets, Hypergraph::Label* edge_labels, const Hypergraph::Label* vertex_labels, std::size_t num_edges) {
    const std::size_t edge = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    if (edge >= num_edges) { return; }

    __shared__ float shared_counts[MAX_LABELS * MAX_CUDA_BLOCK_SIZE];
    float* counts = &shared_counts[static_cast<std::size_t>(threadIdx.x) * MAX_LABELS];

    for (int i = 0; i < MAX_LABELS; ++i) { counts[i] = 0.0f; }

    const std::size_t v_begin = edge_offsets[edge];
    const std::size_t v_end = edge_offsets[edge + 1];
    for (std::size_t idx = v_begin; idx < v_end; ++idx) {
        const auto v = edge_vertices[idx];
        const int label = static_cast<int>(vertex_labels[v]);
        if (label >= 0 && label < MAX_LABELS) { counts[label] += 1.0f; }
    }

    int best_label = edge_labels[edge];
    float best_weight = -1.0f;
    for (int label = 0; label < MAX_LABELS; ++label) {
        const float weight = counts[label];
        if (weight > best_weight) {
            best_weight = weight;
            best_label = label;
        }
    }

    edge_labels[edge] = static_cast<Hypergraph::Label>(best_label);
}

__global__ void update_vertex_labels_kernel(const Hypergraph::EdgeId* vertex_edges,
                                            const std::size_t* vertex_offsets,
                                            const Hypergraph::Label* edge_labels,
                                            Hypergraph::Label* vertex_labels,
                                            unsigned int* change_flags,
                                            std::size_t num_vertices) {
    const std::size_t vertex = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (vertex >= num_vertices) { return; }

    __shared__ float shared_counts[MAX_LABELS * MAX_CUDA_BLOCK_SIZE];
    float* counts = &shared_counts[static_cast<std::size_t>(threadIdx.x) * MAX_LABELS];

    for (int i = 0; i < MAX_LABELS; ++i) { counts[i] = 0.0f; }

    const std::size_t e_begin = vertex_offsets[vertex];
    const std::size_t e_end = vertex_offsets[vertex + 1];
    for (std::size_t idx = e_begin; idx < e_end; ++idx) {
        const auto edge = vertex_edges[idx];
        const int label = static_cast<int>(edge_labels[edge]);
        if (label >= 0 && label < MAX_LABELS) { counts[label] += 1.0f; }
    }

    int best_label = vertex_labels[vertex];
    float best_weight = -1.0f;
    for (int label = 0; label < MAX_LABELS; ++label) {
        const float weight = counts[label];
        if (weight > best_weight) {
            best_weight = weight;
            best_label = label;
        }
    }

    unsigned int changed = 0;
    if (vertex_labels[vertex] != static_cast<Hypergraph::Label>(best_label)) {
        vertex_labels[vertex] = static_cast<Hypergraph::Label>(best_label);
        changed = 1;
    }

    change_flags[vertex] = changed;
}

} // namespace

LabelPropagationCUDA::LabelPropagationCUDA(const CLI::DeviceOptions& device) : LabelPropagationAlgorithm(device) {
    check_cuda(hipGetDevice(&device_id_), "hipGetDevice");
    hipDeviceProp_t prop{};
    check_cuda(hipGetDeviceProperties(&prop, device_id_), "hipGetDeviceProperties");
    max_threads_per_block_ = std::min(prop.maxThreadsPerBlock, MAX_CUDA_BLOCK_SIZE);

    std::cout << "CUDA device: " << prop.name << "\n";
    std::cout << "  Compute capability: " << prop.major << "." << prop.minor << "\n";
    std::cout << "  Multiprocessors: " << prop.multiProcessorCount << "\n";
}

LabelPropagationCUDA::~LabelPropagationCUDA() = default;

void LabelPropagationCUDA::check_cuda(hipError_t err, const char* context) {
    if (err != hipSuccess) { throw std::runtime_error(std::string("CUDA error in ") + context + ": " + hipGetErrorString(err)); }
}

LabelPropagationCUDA::DeviceFlatHypergraph LabelPropagationCUDA::create_device_hypergraph(const Hypergraph& hypergraph) {
    DeviceFlatHypergraph device_hg;
    auto flat_hg = hypergraph.flatten();

    const auto copy_vector = [&](auto& host_vec, auto*& device_ptr) {
        using ValueType = typename std::remove_reference<decltype(host_vec)>::type::value_type;
        if (host_vec.empty()) {
            device_ptr = nullptr;
            return;
        }
        ValueType* raw_ptr = nullptr;
        check_cuda(hipMalloc(reinterpret_cast<void**>(&raw_ptr), host_vec.size() * sizeof(ValueType)), "hipMalloc(flattened data)");
        check_cuda(hipMemcpy(raw_ptr, host_vec.data(), host_vec.size() * sizeof(ValueType), hipMemcpyHostToDevice), "hipMemcpy(flattened data)");
        device_ptr = raw_ptr;
    };

    copy_vector(flat_hg.edge_vertices, device_hg.edge_vertices);
    copy_vector(flat_hg.edge_offsets, device_hg.edge_offsets);
    copy_vector(flat_hg.vertex_edges, device_hg.vertex_edges);
    copy_vector(flat_hg.vertex_offsets, device_hg.vertex_offsets);
    copy_vector(flat_hg.edge_sizes, device_hg.edge_sizes);

    device_hg.num_vertices = flat_hg.num_vertices;
    device_hg.num_edges = flat_hg.num_edges;

    return device_hg;
}

void LabelPropagationCUDA::destroy_device_hypergraph(DeviceFlatHypergraph& flat_hg) {
    if (flat_hg.edge_vertices != nullptr) {
        hipFree(flat_hg.edge_vertices);
        flat_hg.edge_vertices = nullptr;
    }
    if (flat_hg.edge_offsets != nullptr) {
        hipFree(flat_hg.edge_offsets);
        flat_hg.edge_offsets = nullptr;
    }
    if (flat_hg.vertex_edges != nullptr) {
        hipFree(flat_hg.vertex_edges);
        flat_hg.vertex_edges = nullptr;
    }
    if (flat_hg.vertex_offsets != nullptr) {
        hipFree(flat_hg.vertex_offsets);
        flat_hg.vertex_offsets = nullptr;
    }
    if (flat_hg.edge_sizes != nullptr) {
        hipFree(flat_hg.edge_sizes);
        flat_hg.edge_sizes = nullptr;
    }
}

PerformanceMeasurer LabelPropagationCUDA::run(Hypergraph& hypergraph, int max_iterations, double tolerance) {
    std::cout << "Running CUDA label propagation\n";

    PerformanceMeasurer perf;
    const auto overall_start = PerformanceMeasurer::clock::now();

    const std::size_t num_vertices = hypergraph.get_num_vertices();
    const std::size_t num_edges = hypergraph.get_num_edges();

    if (num_vertices == 0 || num_edges == 0) {
        std::cout << "Empty hypergraph detected; nothing to compute.\n";
        perf.set_iterations(0);
        perf.set_total_time(PerformanceMeasurer::clock::now() - overall_start);
        return perf;
    }

    DeviceFlatHypergraph flat_hg{};
    Hypergraph::Label* d_vertex_labels = nullptr;
    Hypergraph::Label* d_edge_labels = nullptr;
    unsigned int* d_change_flags = nullptr;

    auto cleanup = [&]() {
        if (d_vertex_labels) {
            hipFree(d_vertex_labels);
            d_vertex_labels = nullptr;
        }
        if (d_edge_labels) {
            hipFree(d_edge_labels);
            d_edge_labels = nullptr;
        }
        if (d_change_flags) {
            hipFree(d_change_flags);
            d_change_flags = nullptr;
        }
        destroy_device_hypergraph(flat_hg);
    };

    try {
        const auto setup_start = PerformanceMeasurer::clock::now();

        flat_hg = create_device_hypergraph(hypergraph);

        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_vertex_labels), num_vertices * sizeof(Hypergraph::Label)), "hipMalloc(vertex_labels)");
        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_edge_labels), num_edges * sizeof(Hypergraph::Label)), "hipMalloc(edge_labels)");
        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_change_flags), num_vertices * sizeof(unsigned int)), "hipMalloc(change_flags)");

        const auto initial_labels = hypergraph.get_labels();
        check_cuda(hipMemcpy(d_vertex_labels, initial_labels.data(), num_vertices * sizeof(Hypergraph::Label), hipMemcpyHostToDevice), "hipMemcpy(vertex_labels)");
        check_cuda(hipMemset(d_edge_labels, 0, num_edges * sizeof(Hypergraph::Label)), "hipMemset(edge_labels)");

        const auto setup_end = PerformanceMeasurer::clock::now();
        perf.add_moment("setup", setup_end - setup_start);

        const auto iteration_start = PerformanceMeasurer::clock::now();
        int iterations_completed = 0;
        bool converged = false;
        for (int iteration = 0; iteration < max_iterations; ++iteration) {
            const bool iteration_converged = run_iteration_cuda(flat_hg, d_vertex_labels, d_edge_labels, d_change_flags, tolerance);
            if (iteration_converged) {
                std::cout << "Converged after " << iteration + 1 << " iterations\n";
                iterations_completed = iteration + 1;
                converged = true;
                break;
            }
            if ((iteration + 1) % 10 == 0) { std::cout << "Iteration " << iteration + 1 << " completed\n"; }
        }
        if (!converged) { iterations_completed = max_iterations; }

        const auto iteration_end = PerformanceMeasurer::clock::now();
        perf.add_moment("iterations", iteration_end - iteration_start);

        const auto finalize_start = PerformanceMeasurer::clock::now();
        std::vector<Hypergraph::Label> host_labels(num_vertices);
        check_cuda(hipMemcpy(host_labels.data(), d_vertex_labels, num_vertices * sizeof(Hypergraph::Label), hipMemcpyDeviceToHost), "hipMemcpy(host_labels)");
        hypergraph.set_labels(host_labels);
        const auto finalize_end = PerformanceMeasurer::clock::now();
        perf.add_moment("finalize", finalize_end - finalize_start);

        perf.set_iterations(iterations_completed);
        perf.set_total_time(PerformanceMeasurer::clock::now() - overall_start);

        cleanup();
        return perf;
    } catch (...) {
        cleanup();
        throw;
    }
}

bool LabelPropagationCUDA::run_iteration_cuda(
    const DeviceFlatHypergraph& flat_hg, Hypergraph::Label* d_vertex_labels, Hypergraph::Label* d_edge_labels, unsigned int* d_change_flags, double tolerance) {
    if (flat_hg.num_vertices == 0) { return true; }

    int threads = device_.workgroup_size > 0 ? static_cast<int>(device_.workgroup_size) : 256;
    threads = std::max(1, std::min(threads, max_threads_per_block_));

    int pow2_threads = 1;
    while (pow2_threads * 2 <= threads) { pow2_threads *= 2; }
    threads = pow2_threads;

    const dim3 block_dim(static_cast<unsigned int>(threads));
    const dim3 edge_grid_dim(static_cast<unsigned int>((flat_hg.num_edges + block_dim.x - 1) / block_dim.x));

    if (flat_hg.num_edges > 0) {
        update_edge_labels_kernel<<<edge_grid_dim, block_dim>>>(flat_hg.edge_vertices, flat_hg.edge_offsets, d_edge_labels, d_vertex_labels, flat_hg.num_edges);
        check_cuda(hipGetLastError(), "update_edge_labels_kernel");
    }

    const dim3 vertex_grid_dim(static_cast<unsigned int>((flat_hg.num_vertices + block_dim.x - 1) / block_dim.x));
    if (flat_hg.num_vertices > 0) {
        update_vertex_labels_kernel<<<vertex_grid_dim, block_dim>>>(flat_hg.vertex_edges, flat_hg.vertex_offsets, d_edge_labels, d_vertex_labels, d_change_flags, flat_hg.num_vertices);
        check_cuda(hipGetLastError(), "update_vertex_labels_kernel");
    }

    check_cuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    thrust::device_ptr<unsigned int> changes_begin(d_change_flags);
    const unsigned int change_count = thrust::reduce(thrust::device, changes_begin, changes_begin + flat_hg.num_vertices, 0u);

    const double change_ratio = static_cast<double>(change_count) / static_cast<double>(flat_hg.num_vertices);
    return change_ratio < tolerance;
}
