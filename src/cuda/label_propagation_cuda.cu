#include "hip/hip_runtime.h"
#include "label_propagation_cuda.hpp"
#include <algorithm>
#include <chrono>
#include <iostream>
#include <stdexcept>
#include <type_traits>
#include <vector>

namespace {

constexpr int MAX_LABELS = 10; // must cover possible label values
constexpr int MAX_CUDA_BLOCK_SIZE = 1024;

__global__ void update_edge_labels_kernel(const Hypergraph::VertexId* edge_vertices,
                                          const std::size_t* edge_offsets,
                                          Hypergraph::Label* edge_labels,
                                          const Hypergraph::Label* vertex_labels,
                                          unsigned long long* changes,
                                          std::size_t num_edges) {
    const std::size_t edge = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    if (edge >= num_edges) { return; }

    if (changes != nullptr && threadIdx.x == 0) { *changes = 0; }

    __shared__ float shared_counts[MAX_LABELS * MAX_CUDA_BLOCK_SIZE];
    float* counts = &shared_counts[static_cast<std::size_t>(threadIdx.x) * MAX_LABELS];

    for (int i = 0; i < MAX_LABELS; ++i) { counts[i] = 0.0f; }

    const std::size_t v_begin = edge_offsets[edge];
    const std::size_t v_end = edge_offsets[edge + 1];
    for (std::size_t idx = v_begin; idx < v_end; ++idx) {
        const auto v = edge_vertices[idx];
        const int label = static_cast<int>(vertex_labels[v]);
        if (label >= 0 && label < MAX_LABELS) { counts[label] += 1.0f; }
    }

    int best_label = edge_labels[edge];
    float best_weight = -1.0f;
    for (int label = 0; label < MAX_LABELS; ++label) {
        const float weight = counts[label];
        if (weight > best_weight) {
            best_weight = weight;
            best_label = label;
        }
    }

    edge_labels[edge] = static_cast<Hypergraph::Label>(best_label);
}

__global__ void update_vertex_labels_kernel(const Hypergraph::EdgeId* vertex_edges,
                                            const std::size_t* vertex_offsets,
                                            const Hypergraph::Label* edge_labels,
                                            Hypergraph::Label* vertex_labels,
                                            unsigned long long* changes,
                                            std::size_t num_vertices) {
    const std::size_t vertex = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    __shared__ float shared_counts[MAX_LABELS * MAX_CUDA_BLOCK_SIZE];
    __shared__ unsigned int change_buffer[MAX_CUDA_BLOCK_SIZE];
    float* counts = &shared_counts[static_cast<std::size_t>(threadIdx.x) * MAX_LABELS];

    for (int i = 0; i < MAX_LABELS; ++i) { counts[i] = 0.0f; }

    if (vertex < num_vertices) {
        const std::size_t e_begin = vertex_offsets[vertex];
        const std::size_t e_end = vertex_offsets[vertex + 1];
        for (std::size_t idx = e_begin; idx < e_end; ++idx) {
            const auto edge = vertex_edges[idx];
            const int label = static_cast<int>(edge_labels[edge]);
            if (label >= 0 && label < MAX_LABELS) { counts[label] += 1.0f; }
        }
    }

    int best_label = vertex < num_vertices ? vertex_labels[vertex] : 0;
    float best_weight = -1.0f;
    for (int label = 0; label < MAX_LABELS; ++label) {
        const float weight = counts[label];
        if (weight > best_weight) {
            best_weight = weight;
            best_label = label;
        }
    }

    unsigned int changed = 0;
    if (vertex < num_vertices) {
        if (vertex_labels[vertex] != static_cast<Hypergraph::Label>(best_label)) {
            vertex_labels[vertex] = static_cast<Hypergraph::Label>(best_label);
            changed = 1;
        }
    }

    change_buffer[threadIdx.x] = changed;
    __syncthreads();

    // Reduce per-thread change flags to a single block contribution
    for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) { change_buffer[threadIdx.x] += change_buffer[threadIdx.x + offset]; }
        __syncthreads();
    }

    if (threadIdx.x == 0 && change_buffer[0] > 0) { atomicAdd(changes, static_cast<unsigned long long>(change_buffer[0])); }
}

} // namespace

LabelPropagationCUDA::LabelPropagationCUDA(const CLI::DeviceOptions& device) : LabelPropagationAlgorithm(device) {
    check_cuda(hipGetDevice(&device_id_), "hipGetDevice");
    hipDeviceProp_t prop{};
    check_cuda(hipGetDeviceProperties(&prop, device_id_), "hipGetDeviceProperties");
    max_threads_per_block_ = std::min(prop.maxThreadsPerBlock, MAX_CUDA_BLOCK_SIZE);

    std::cout << "CUDA device: " << prop.name << "\n";
    std::cout << "  Compute capability: " << prop.major << "." << prop.minor << "\n";
    std::cout << "  Multiprocessors: " << prop.multiProcessorCount << "\n";
}

LabelPropagationCUDA::~LabelPropagationCUDA() = default;

void LabelPropagationCUDA::check_cuda(hipError_t err, const char* context) {
    if (err != hipSuccess) { throw std::runtime_error(std::string("CUDA error in ") + context + ": " + hipGetErrorString(err)); }
}

LabelPropagationCUDA::DeviceFlatHypergraph LabelPropagationCUDA::create_device_hypergraph(const Hypergraph& hypergraph) {
    DeviceFlatHypergraph device_hg;
    auto flat_hg = hypergraph.flatten();

    const auto copy_vector = [&](auto& host_vec, auto*& device_ptr) {
        using ValueType = typename std::remove_reference<decltype(host_vec)>::type::value_type;
        if (host_vec.empty()) {
            device_ptr = nullptr;
            return;
        }
        ValueType* raw_ptr = nullptr;
        check_cuda(hipMalloc(reinterpret_cast<void**>(&raw_ptr), host_vec.size() * sizeof(ValueType)), "hipMalloc(flattened data)");
        check_cuda(hipMemcpy(raw_ptr, host_vec.data(), host_vec.size() * sizeof(ValueType), hipMemcpyHostToDevice), "hipMemcpy(flattened data)");
        device_ptr = raw_ptr;
    };

    copy_vector(flat_hg.edge_vertices, device_hg.edge_vertices);
    copy_vector(flat_hg.edge_offsets, device_hg.edge_offsets);
    copy_vector(flat_hg.vertex_edges, device_hg.vertex_edges);
    copy_vector(flat_hg.vertex_offsets, device_hg.vertex_offsets);
    copy_vector(flat_hg.edge_sizes, device_hg.edge_sizes);

    device_hg.num_vertices = flat_hg.num_vertices;
    device_hg.num_edges = flat_hg.num_edges;

    return device_hg;
}

void LabelPropagationCUDA::destroy_device_hypergraph(DeviceFlatHypergraph& flat_hg) {
    if (flat_hg.edge_vertices != nullptr) {
        hipFree(flat_hg.edge_vertices);
        flat_hg.edge_vertices = nullptr;
    }
    if (flat_hg.edge_offsets != nullptr) {
        hipFree(flat_hg.edge_offsets);
        flat_hg.edge_offsets = nullptr;
    }
    if (flat_hg.vertex_edges != nullptr) {
        hipFree(flat_hg.vertex_edges);
        flat_hg.vertex_edges = nullptr;
    }
    if (flat_hg.vertex_offsets != nullptr) {
        hipFree(flat_hg.vertex_offsets);
        flat_hg.vertex_offsets = nullptr;
    }
    if (flat_hg.edge_sizes != nullptr) {
        hipFree(flat_hg.edge_sizes);
        flat_hg.edge_sizes = nullptr;
    }
}

int LabelPropagationCUDA::run(Hypergraph& hypergraph, int max_iterations, double tolerance) {
    std::cout << "Running CUDA label propagation\n";

    DeviceFlatHypergraph flat_hg{};
    Hypergraph::Label* d_vertex_labels = nullptr;
    Hypergraph::Label* d_edge_labels = nullptr;
    unsigned long long* d_changes = nullptr;


    try {
        auto start = std::chrono::high_resolution_clock::now();
        flat_hg = create_device_hypergraph(hypergraph);

        const std::size_t num_vertices = hypergraph.get_num_vertices();
        const std::size_t num_edges = hypergraph.get_num_edges();

        if (num_vertices == 0 || num_edges == 0) {
            std::cout << "Empty hypergraph detected; nothing to compute.\n";
            destroy_device_hypergraph(flat_hg);
            return 0;
        }

        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_vertex_labels), num_vertices * sizeof(Hypergraph::Label)), "hipMalloc(vertex_labels)");
        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_edge_labels), num_edges * sizeof(Hypergraph::Label)), "hipMalloc(edge_labels)");
        check_cuda(hipMalloc(reinterpret_cast<void**>(&d_changes), sizeof(unsigned long long)), "hipMalloc(changes)");

        const auto initial_labels = hypergraph.get_labels();
        check_cuda(hipMemcpy(d_vertex_labels, initial_labels.data(), num_vertices * sizeof(Hypergraph::Label), hipMemcpyHostToDevice), "hipMemcpy(vertex_labels)");
        check_cuda(hipMemset(d_edge_labels, 0, num_edges * sizeof(Hypergraph::Label)), "hipMemset(edge_labels)");
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        std::cout << "Data transfer to GPU took " << duration.count() << " ms\n";
        int iteration = 0;
        for (iteration = 0; iteration < max_iterations; ++iteration) {
            const bool converged = run_iteration_cuda(flat_hg, d_vertex_labels, d_edge_labels, d_changes, tolerance);
            if (converged) {
                std::cout << "Converged after " << iteration + 1 << " iterations\n";
                break;
            }
            if ((iteration + 1) % 10 == 0) { std::cout << "Iteration " << iteration + 1 << " completed\n"; }
        }

        std::vector<Hypergraph::Label> host_labels(num_vertices);
        check_cuda(hipMemcpy(host_labels.data(), d_vertex_labels, num_vertices * sizeof(Hypergraph::Label), hipMemcpyDeviceToHost), "hipMemcpy(host_labels)");
        hypergraph.set_labels(host_labels);

        destroy_device_hypergraph(flat_hg);
        if (d_vertex_labels) { hipFree(d_vertex_labels); }
        if (d_edge_labels) { hipFree(d_edge_labels); }
        if (d_changes) { hipFree(d_changes); }

        return iteration + 1;
    } catch (...) {
        if (d_vertex_labels) { hipFree(d_vertex_labels); }
        if (d_edge_labels) { hipFree(d_edge_labels); }
        if (d_changes) { hipFree(d_changes); }
        destroy_device_hypergraph(flat_hg);
        throw;
    }
}

bool LabelPropagationCUDA::run_iteration_cuda(
    const DeviceFlatHypergraph& flat_hg, Hypergraph::Label* d_vertex_labels, Hypergraph::Label* d_edge_labels, unsigned long long* d_changes, double tolerance) {
    if (flat_hg.num_vertices == 0) { return true; }

    int threads = device_.workgroup_size > 0 ? static_cast<int>(device_.workgroup_size) : 256;
    threads = std::max(1, std::min(threads, max_threads_per_block_));

    int pow2_threads = 1;
    while (pow2_threads * 2 <= threads) { pow2_threads *= 2; }
    threads = pow2_threads;

    const dim3 block_dim(static_cast<unsigned int>(threads));
    const dim3 edge_grid_dim(static_cast<unsigned int>((flat_hg.num_edges + block_dim.x - 1) / block_dim.x));

    if (flat_hg.num_edges > 0) {
        update_edge_labels_kernel<<<edge_grid_dim, block_dim>>>(flat_hg.edge_vertices, flat_hg.edge_offsets, d_edge_labels, d_vertex_labels, d_changes, flat_hg.num_edges);
        check_cuda(hipGetLastError(), "update_edge_labels_kernel");
    }

    const dim3 vertex_grid_dim(static_cast<unsigned int>((flat_hg.num_vertices + block_dim.x - 1) / block_dim.x));
    if (flat_hg.num_vertices > 0) {
        update_vertex_labels_kernel<<<vertex_grid_dim, block_dim>>>(flat_hg.vertex_edges, flat_hg.vertex_offsets, d_edge_labels, d_vertex_labels, d_changes, flat_hg.num_vertices);
        check_cuda(hipGetLastError(), "update_vertex_labels_kernel");
    }

    check_cuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    unsigned long long change_count = 0;
    check_cuda(hipMemcpy(&change_count, d_changes, sizeof(unsigned long long), hipMemcpyDeviceToHost), "hipMemcpy(changes)");

    const double change_ratio = static_cast<double>(change_count) / static_cast<double>(flat_hg.num_vertices);
    return change_ratio < tolerance;
}
